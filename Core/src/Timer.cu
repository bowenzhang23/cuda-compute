
#include "Error.cuh"
#include "Timer.cuh"

void Timer::Tick(const hipStream_t stream)
{
    m_isRecording = !m_isRecording;
    if (m_isRecording) // start
    {
        m_elapsedTime = 0.f;
        CUDA_CHECK(hipEventRecord(m_begin, stream));
    } else // end
    {
        CUDA_CHECK(hipEventRecord(m_end, stream));
        CUDA_CHECK(hipEventSynchronize(m_begin));
        CUDA_CHECK(hipEventSynchronize(m_end));
        CUDA_CHECK(hipEventElapsedTime(&m_elapsedTime, m_begin, m_end));
    }
}

Timer::Timer()
    : m_isRecording(false), m_elapsedTime(0.f), m_begin(), m_end()
{
    CUDA_CHECK(hipEventCreate(&m_begin));
    CUDA_CHECK(hipEventCreate(&m_end));
}