#include "hip/hip_runtime.h"
#include "DeviceManager.cuh"
#include "Error.cuh"
#include "Stream.cuh"

Stream::Stream()
{
    CUDA_CHECK(hipStreamCreate(&m_stream));
    Sync();
}

Stream::Stream(unsigned flags)
{
    CUDA_CHECK(hipStreamCreateWithFlags(&m_stream, flags));
    Sync();
}

Stream::Stream(unsigned flags, int priority)
{
    CUDA_CHECK(hipStreamCreateWithPriority(&m_stream, flags, priority));
    Sync();
}

Stream::~Stream()
{
    Sync();
    CUDA_CHECK(hipStreamDestroy(m_stream));
}

void Stream::Sync() { CUDA_CHECK(hipStreamSynchronize(m_stream)); }

Stream* Stream::Create() { return new Stream(); }

Stream* Stream::CreateNonBlocking()
{
    return new Stream(hipStreamNonBlocking);
}

Stream* Stream::CreateNonBlocking(int p)
{
    const auto& device = DeviceManager::Instance().CurrentDevice();
    auto        pg     = device.StreamPriorityGreatest();
    auto        pl     = device.StreamPriorityLeast();
    p += pg;
    p = min(p, pl);

    fprintf(stdout, "Creating non-blocking stream with priority %d\n", p);
    return new Stream(hipStreamNonBlocking, p);
}
