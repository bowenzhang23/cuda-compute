#include "hip/hip_runtime.h"
#include "Matrix.cuh"
#include "Vector.cuh"
#include "nanobind/nanobind.h"
#include "nanobind/stl/array.h"
#include "nanobind/stl/pair.h"
#include "nanobind/stl/string.h"
#include "nanobind/stl/vector.h"

namespace nb = nanobind;
using namespace nb::literals;

using Vectorf = Vector<float>;
using Vectori = Vector<int>;

using Matrixf = Matrix<float>;
using Matrixi = Matrix<int>;

template <typename T>
std::pair<T, int> ToPair(const ValueIndex<T>& vi)
{
    return std::make_pair(vi.val, vi.idx);
}

template <typename Tnb, typename Tcd, typename Tcd_int, typename T>
[[maybe_unused]] Tnb& add_arithmetic(Tnb& cls)
{
    cls.def("__pos__", [](const Tcd& a) { return +a; });
    cls.def("__neg__", [](const Tcd& a) { return -a; });
    cls.def("__pow__", [](const Tcd& a, const T b) {
        return Power((T) 1, a, b, a, (T) 0);
    });
    cls.def("__add__", [](const Tcd& a, const T b) { return a + b; });
    cls.def("__radd__", [](const Tcd& a, const T b) { return b + a; });
    cls.def("__sub__", [](const Tcd& a, const T b) { return a - b; });
    cls.def("__rsub__", [](const Tcd& a, const T b) { return b - a; });
    cls.def("__mul__", [](const Tcd& a, const T b) { return a * b; });
    cls.def("__rmul__", [](const Tcd& a, const T b) { return b * a; });
    cls.def("__truediv__", [](const Tcd& a, const T b) { return a / b; });
    cls.def("__rtruediv__", [](const Tcd& a, const T b) { return b / a; });
    cls.def("__eq__", [](const Tcd& a, const T b) { return a == b; });
    cls.def("__req__", [](const Tcd& a, const T b) { return b == a; });
    cls.def("__ne__", [](const Tcd& a, const T b) { return a != b; });
    cls.def("__rne__", [](const Tcd& a, const T b) { return b != a; });
    cls.def("__gt__", [](const Tcd& a, const T b) { return a > b; });
    cls.def("__rgt__", [](const Tcd& a, const T b) { return b > a; });
    cls.def("__ge__", [](const Tcd& a, const T b) { return a >= b; });
    cls.def("__rge__", [](const Tcd& a, const T b) { return b >= a; });
    cls.def("__lt__", [](const Tcd& a, const T b) { return a < b; });
    cls.def("__rlt__", [](const Tcd& a, const T b) { return b < a; });
    cls.def("__le__", [](const Tcd& a, const T b) { return a <= b; });
    cls.def("__rle__", [](const Tcd& a, const T b) { return b <= a; });
    cls.def("__add__", [](const Tcd& a, const Tcd& b) { return a + b; });
    cls.def("__sub__", [](const Tcd& a, const Tcd& b) { return a - b; });
    cls.def("__mul__", [](const Tcd& a, const Tcd& b) { return a * b; });
    cls.def("__truediv__", [](const Tcd& a, const Tcd& b) { return a / b; });
    cls.def("__eq__", [](const Tcd& a, const Tcd& b) { return a == b; });
    cls.def("__ne__", [](const Tcd& a, const Tcd& b) { return a != b; });
    cls.def("__gt__", [](const Tcd& a, const Tcd& b) { return a > b; });
    cls.def("__ge__", [](const Tcd& a, const Tcd& b) { return a >= b; });
    cls.def("__lt__", [](const Tcd& a, const Tcd& b) { return a < b; });
    cls.def("__le__", [](const Tcd& a, const Tcd& b) { return a <= b; });

    return cls;
}

template <typename Tnb, typename Tcd, typename T>
[[maybe_unused]] Tnb& add_binary(Tnb& m)
{
    m.def("max", [](const Tcd& a, const T b) { return max(a, b); });
    m.def("max", [](const T b, const Tcd& a) { return max(b, a); });
    m.def("max", [](const Tcd& a, const Tcd& b) { return max(a, b); });
    m.def("min", [](const Tcd& a, const T b) { return min(a, b); });
    m.def("min", [](const T b, const Tcd& a) { return min(b, a); });
    m.def("min", [](const Tcd& a, const Tcd& b) { return min(a, b); });

    return m;
}

NB_MODULE(cuda_compute, m)
{
    nb::class_<Device>(m, "DeviceBase")
        .def(nb::init<int>())
        .def("id", &Device::ID);

    auto vf = nb::class_<Vectorf>(m, "VectorfBase");
    vf.def(nb::init<unsigned long>());
    vf.def(nb::init<const std::vector<float>&, unsigned long>());
    vf.def("cpu", &Vectorf::ToCPU);
    vf.def("shape", &Vectorf::Shape);
    vf.def("sum", &Vectorf::Sum);
    vf.def("mean", &Vectorf::Mean);
    vf.def("max", [](const Vectorf& a) { return ToPair(a.Max()); });
    vf.def("min", [](const Vectorf& a) { return ToPair(a.Min()); });
    vf.def("reversed", &Vectorf::Reversed);

    auto vi = nb::class_<Vectori>(m, "VectoriBase");
    vi.def(nb::init<unsigned long>());
    vi.def(nb::init<const std::vector<int>&, unsigned long>());
    vi.def("cpu", &Vectori::ToCPU);
    vi.def("shape", &Vectori::Shape);
    vi.def("sum", &Vectori::Sum);
    vi.def("mean", &Vectori::Mean);
    vi.def("max", [](const Vectori& a) { return ToPair(a.Max()); });
    vi.def("min", [](const Vectori& a) { return ToPair(a.Min()); });
    vi.def("reversed", &Vectori::Reversed);

    auto mf = nb::class_<Matrixf>(m, "MatrixfBase");
    mf.def(nb::init<unsigned long, unsigned long>());
    mf.def(nb::init<const std::vector<float>&, unsigned long, unsigned long>());
    mf.def("cpu", &Matrixf::ToCPU);
    mf.def("shape", &Matrixf::Shape);
    mf.def("transpose", &Matrixf::Transpose);

    auto mi = nb::class_<Matrixi>(m, "MatrixiBase");
    mi.def(nb::init<unsigned long, unsigned long>());
    mi.def(nb::init<const std::vector<int>&, unsigned long, unsigned long>());
    mi.def("cpu", &Matrixi::ToCPU);
    mi.def("shape", &Matrixi::Shape);
    mi.def("transpose", &Matrixi::Transpose);

    add_arithmetic<decltype(vf), Vectorf, Vectori, float>(vf);
    add_arithmetic<decltype(vi), Vectori, Vectori, int>(vi);
    add_arithmetic<decltype(mf), Matrixf, Matrixi, float>(mf);
    add_arithmetic<decltype(mi), Matrixi, Matrixi, int>(mi);

    add_binary<decltype(m), Vectorf, float>(m);
    add_binary<decltype(m), Vectori, int>(m);
    add_binary<decltype(m), Matrixf, float>(m);
    add_binary<decltype(m), Matrixi, int>(m);

    m.def("use_device",
          [](int id) { DeviceManager::Instance().UseDevice(id); });
    m.def("current_device",
          []() { return DeviceManager::Instance().CurrentDevice(); });
    m.def("device_query",
          []() { return DeviceManager::Instance().ToString(); });
    m.def("timer_start", []() { Timer::Instance().Tick(); });
    m.def("timer_stop", []() {
        Timer::Instance().Tick();
        return Timer::Instance().ElapsedTime();
    });

    m.def("gemm", &MatMul<float>, "a"_a, "b"_a);
    m.def("gemm", &MatMul<int>, "a"_a, "b"_a);
    m.def("inner", &Inner<float>, "a"_a, "b"_a);
    m.def("inner", &Inner<int>, "a"_a, "b"_a);
}
